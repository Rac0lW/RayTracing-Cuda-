#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "vec3.h"
#include "color.h"
#include "ray.h"
#include "hitable_list.h"
#include "camera.h"

#include "error_check.h"
#include "memery_allocate.h"
#include "hitable.h"
#include "hitable_list.h"
#include "sphere.h"
#include "render.h"
#include "error_check.h"
#include "random_tools.h"



int main() {
    int nx = 1200;
    int ny = 600;
    int ns = 100;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init << <1, 1 >> > (d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hitable** d_list;
    int num_hitables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void**)&d_list, num_hitables * sizeof(hitable*)));//TODO:

    hitable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));

    create_world << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    render_init << <blocks, threads >> > (nx, ny,d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render << <blocks, threads >> > (fb, nx, ny,ns,d_camera,d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());

    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}